#include "hip/hip_runtime.h"
#include <stdio.h>

#define NUM_BLOCKS 16

// number of threads per block
#define BLOCK_WIDTH 1

__global__ void hello() {
  printf("hello, I'm a thread in block %d\n", blockIdx.x);
}

int main(int argc, char** argv) {
  hello<NUM_BLOCKS, BLOCK_WIDTH>>();

  // force the printf()s to flush
  hipDeviceSynchronize();

  printf("done!\n");

  return 0;
}
